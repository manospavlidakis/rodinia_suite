#include "hip/hip_runtime.h"
/*-----------------------------------------------------------
 ** gaussian.cu -- The program is to solve a linear system Ax = b
 **   by using Gaussian Elimination. The algorithm on page 101
 **   ("Foundations of Parallel Programming") is used.
 **   The sequential version is gaussian.c.  This parallel
 **   implementation converts three independent for() loops
 **   into three Fans.  Use the data file ge_3.dat to verify
 **   the correction of the output.
 **
 ** Written by Andreas Kura, 02/15/95
 ** Modified by Chong-wei Xu, 04/20/95
 ** Modified by Chris Gregg for CUDA, 07/20/2009
 **-----------------------------------------------------------
 */
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

std::chrono::high_resolution_clock::time_point s_compute;
std::chrono::high_resolution_clock::time_point e_compute;
std::chrono::high_resolution_clock::time_point start_warmup;
std::chrono::high_resolution_clock::time_point end_warmup;
#define WARMUP
#ifdef RD_WG_SIZE_0_0
#define MAXBLOCKSIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
#define MAXBLOCKSIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
#define MAXBLOCKSIZE RD_WG_SIZE
#else
#define MAXBLOCKSIZE 512
#endif

// 2D defines. Go from specific to general
#ifdef RD_WG_SIZE_1_0
#define BLOCK_SIZE_XY RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
#define BLOCK_SIZE_XY RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
#define BLOCK_SIZE_XY RD_WG_SIZE
#else
#define BLOCK_SIZE_XY 4
#endif

int Size;
float *a, *b, *finalVec;
float *m;

FILE *fp;

void InitProblemOnce(char *filename);
void InitPerRun();
void ForwardSub();
void BackSub();
__global__ void Fan1(float *m, float *a, int Size, int t);
__global__ void Fan2(float *m, float *a, float *b, int Size, int j1, int t);
void InitMat(float *ary, int nrow, int ncol);
void InitAry(float *ary, int ary_size);
void PrintMat(float *ary, int nrow, int ncolumn);
void PrintAry(float *ary, int ary_size);
void PrintDeviceProperties();
void checkCUDAError(const char *msg);
void VerifyResult();

unsigned int totalKernelTime = 0;

// create both matrix and right hand side, Ke Wang 2013/08/12 11:51:06
void create_matrix(float *m, int size) {
  int i, j;
  float lamda = -0.01;
  float coe[2 * size - 1];
  float coe_i = 0.0;

  for (i = 0; i < size; i++) {
    coe_i = 10 * exp(lamda * i);
    j = size - 1 + i;
    coe[j] = coe_i;
    j = size - 1 - i;
    coe[j] = coe_i;
  }

  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      m[i * size + j] = coe[size - 1 - i + j];
    }
  }
}

int main(int argc, char *argv[]) {
  auto start = std::chrono::high_resolution_clock::now();
  auto start_0 = std::chrono::high_resolution_clock::now();

  int i, j;
  char flag;
  if (argc < 2) {
    printf("Usage: gaussian -f filename / -s size [-q]\n\n");
    printf("-q (quiet) suppresses printing the matrix and result values.\n");
    printf("-f (filename) path of input file\n");
    printf(
        "-s (size) size of matrix. Create matrix and rhs in this program \n");
    printf(
        "The first line of the file contains the dimension of the matrix, n.");
    printf("The second line of the file is a newline.\n");
    printf("The next n lines contain n tab separated values for the matrix.");
    printf("The next line of the file is a newline.\n");
    printf("The next line of the file is a 1xn vector with tab separated "
           "values.\n");
    printf("The next line of the file is a newline. (optional)\n");
    printf("The final line of the file is the pre-computed solution. "
           "(optional)\n");
    printf("Example: matrix4.txt:\n");
    printf("4\n");
    printf("\n");
    printf("-0.6	-0.5	0.7	0.3\n");
    printf("-0.3	-0.9	0.3	0.7\n");
    printf("-0.4	-0.5	-0.3	-0.8\n");
    printf("0.0	-0.1	0.2	0.9\n");
    printf("\n");
    printf("-0.85	-0.68	0.24	-0.53\n");
    printf("\n");
    printf("0.7	0.0	-0.4	-0.5\n");
    exit(0);
  }

  for (i = 1; i < argc; i++) {
    if (argv[i][0] == '-') { // flag
      flag = argv[i][1];
      switch (flag) {
      case 's': // platform
        i++;
        Size = atoi(argv[i]);

        a = (float *)malloc(Size * Size * sizeof(float));
        create_matrix(a, Size);

        b = (float *)malloc(Size * sizeof(float));
        for (j = 0; j < Size; j++)
          b[j] = 1.0;

        m = (float *)malloc(Size * Size * sizeof(float));
        break;
      case 'f': // platform
        i++;
        printf("Read file from %s \n", argv[i]);
        InitProblemOnce(argv[i]);
        break;
      }
    }
  }

  finalVec = (float *)malloc(Size * sizeof(float));
  // InitProblemOnce(filename);
  InitPerRun();
  auto end_0 = std::chrono::high_resolution_clock::now();

#ifdef WARMUP
  start_warmup = std::chrono::high_resolution_clock::now();
  // Warmup
  char *warm;
  hipMalloc((void **)&warm, sizeof(char));
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  end_warmup = std::chrono::high_resolution_clock::now();
#endif

  s_compute = std::chrono::high_resolution_clock::now();

  // run kernels
  ForwardSub();

#ifdef INPUT
  printf("Matrix m is: \n");
  PrintMat(m, Size, Size);

  printf("Matrix a is: \n");
  PrintMat(a, Size, Size);

  printf("Array b is: \n");
  PrintAry(b, Size);
#endif

  BackSub();

#ifdef OUTPUT
  printf("The final solution is: \n");
  PrintAry(finalVec, Size);
//  VerifyResult();
#endif
  e_compute = std::chrono::high_resolution_clock::now();

  free(m);
  free(a);
  free(b);
  free(finalVec);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> elapsed_milli_0 = end_0 - start_0;
  std::cerr << "Init time: " << elapsed_milli_0.count() << " ms" << std::endl;

  std::chrono::duration<double, std::milli> compute_milli =
      e_compute - s_compute;
  std::cerr << "Computation: " << compute_milli.count() << " ms" << std::endl;

  std::chrono::duration<double, std::milli> elapsed_milli = end - start;
  std::cerr << "Elapsed time: " << elapsed_milli.count() << " ms" << std::endl;
#ifdef WARMUP
  std::chrono::duration<double, std::milli> elapsed_milli_warmup =
      end_warmup - start_warmup;
  std::cerr << "Warmup time: " << elapsed_milli_warmup.count() << " ms"
            << std::endl;
  // free warmup
  hipFree(warm);
#endif
}
/*------------------------------------------------------
 ** PrintDeviceProperties
 **-----------------------------------------------------
 */
void PrintDeviceProperties() {
  hipDeviceProp_t deviceProp;
  int nDevCount = 0;

  hipGetDeviceCount(&nDevCount);
  printf("Total Device found: %d", nDevCount);
  for (int nDeviceIdx = 0; nDeviceIdx < nDevCount; ++nDeviceIdx) {
    memset(&deviceProp, 0, sizeof(deviceProp));
    if (hipSuccess == hipGetDeviceProperties(&deviceProp, nDeviceIdx)) {
      printf("\nDevice Name \t\t - %s ", deviceProp.name);
      printf("\n**************************************");
      printf("\nTotal Global Memory\t\t\t - %lu KB",
             deviceProp.totalGlobalMem / 1024);
      printf("\nShared memory available per block \t - %lu KB",
             deviceProp.sharedMemPerBlock / 1024);
      printf("\nNumber of registers per thread block \t - %d",
             deviceProp.regsPerBlock);
      printf("\nWarp size in threads \t\t\t - %d", deviceProp.warpSize);
      printf("\nMemory Pitch \t\t\t\t - %zu bytes", deviceProp.memPitch);
      printf("\nMaximum threads per block \t\t - %d",
             deviceProp.maxThreadsPerBlock);
      printf("\nMaximum Thread Dimension (block) \t - %d %d %d",
             deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
             deviceProp.maxThreadsDim[2]);
      printf("\nMaximum Thread Dimension (grid) \t - %d %d %d",
             deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
             deviceProp.maxGridSize[2]);
      printf("\nTotal constant memory \t\t\t - %zu bytes",
             deviceProp.totalConstMem);
      printf("\nCUDA ver \t\t\t\t - %d.%d", deviceProp.major, deviceProp.minor);
      printf("\nClock rate \t\t\t\t - %d KHz", deviceProp.clockRate);
      printf("\nTexture Alignment \t\t\t - %zu bytes",
             deviceProp.textureAlignment);
      printf("\nNumber of Multi processors \t\t - %d\n\n",
             deviceProp.multiProcessorCount);
    } else
      printf("\n%s", hipGetErrorString(hipGetLastError()));
  }
}

/*------------------------------------------------------
 ** InitProblemOnce -- Initialize all of matrices and
 ** vectors by opening a data file specified by the user.
 **
 ** We used dynamic array *a, *b, and *m to allocate
 ** the memory storages.
 **------------------------------------------------------
 */
void InitProblemOnce(char *filename) {
  // char *filename = argv[1];

  // printf("Enter the data file name: ");
  // scanf("%s", filename);
  // printf("The file name is: %s\n", filename);

  fp = fopen(filename, "r");

  fscanf(fp, "%d", &Size);

  a = (float *)malloc(Size * Size * sizeof(float));

  InitMat(a, Size, Size);
  // printf("The input matrix a is:\n");
  // PrintMat(a, Size, Size);
  b = (float *)malloc(Size * sizeof(float));

  InitAry(b, Size);
  // printf("The input array b is:\n");
  // PrintAry(b, Size);

  m = (float *)malloc(Size * Size * sizeof(float));
}

/*------------------------------------------------------
 ** InitPerRun() -- Initialize the contents of the
 ** multipier matrix **m
 **------------------------------------------------------
 */
void InitPerRun() {
  int i;
  for (i = 0; i < Size * Size; i++)
    *(m + i) = 0.0;
}

/*-------------------------------------------------------
 ** Fan1() -- Calculate multiplier matrix
 ** Pay attention to the index.  Index i give the range
 ** which starts from 0 to range-1.  The real values of
 ** the index should be adjust and related with the value
 ** of t which is defined on the ForwardSub().
 **-------------------------------------------------------
 */
__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t) {
  // if(hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x >= Size-1-t) printf(".");
  // printf("blockIDx.x:%d,hipThreadIdx_x:%d,Size:%d,t:%d,Size-1-t:%d\n",hipBlockIdx_x,hipThreadIdx_x,Size,t,Size-1-t);

  if (hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x >= Size - 1 - t)
    return;
  *(m_cuda + Size * (hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x + t + 1) +
    t) =
      *(a_cuda +
        Size * (hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x + t + 1) + t) /
      *(a_cuda + Size * t + t);
}

/*-------------------------------------------------------
 ** Fan2() -- Modify the matrix A into LUD
 **-------------------------------------------------------
 */

__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda, int Size,
                     int j1, int t) {
  if (hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x >= Size - 1 - t)
    return;
  if (hipThreadIdx_y + hipBlockIdx_y * hipBlockDim_y >= Size - t)
    return;

  int xidx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
  int yidx = hipBlockIdx_y * hipBlockDim_y + hipThreadIdx_y;
  // printf("hipBlockIdx_x:%d,hipThreadIdx_x:%d,hipBlockIdx_y:%d,hipThreadIdx_y:%d,hipBlockDim_x:%d,hipBlockDim_y:%d\n",hipBlockIdx_x,hipThreadIdx_x,hipBlockIdx_y,hipThreadIdx_y,hipBlockDim_x,hipBlockDim_y);

  a_cuda[Size * (xidx + 1 + t) + (yidx + t)] -=
      m_cuda[Size * (xidx + 1 + t) + t] * a_cuda[Size * t + (yidx + t)];
  // a_cuda[xidx+1+t][yidx+t] -= m_cuda[xidx+1+t][t] * a_cuda[t][yidx+t];
  if (yidx == 0) {
    // printf("hipBlockIdx_x:%d,hipThreadIdx_x:%d,hipBlockIdx_y:%d,hipThreadIdx_y:%d,hipBlockDim_x:%d,hipBlockDim_y:%d\n",hipBlockIdx_x,hipThreadIdx_x,hipBlockIdx_y,hipThreadIdx_y,hipBlockDim_x,hipBlockDim_y);
    // printf("xidx:%d,yidx:%d\n",xidx,yidx);
    b_cuda[xidx + 1 + t] -=
        m_cuda[Size * (xidx + 1 + t) + (yidx + t)] * b_cuda[t];
  }
}
/*------------------------------------------------------
 ** ForwardSub() -- Forward substitution of Gaussian
 ** elimination.
 **------------------------------------------------------
 */
void ForwardSub() {
  int t;
  float *m_cuda, *a_cuda, *b_cuda;

  // allocate memory on GPU
  hipMalloc((void **)&m_cuda, Size * Size * sizeof(float));

  hipMalloc((void **)&a_cuda, Size * Size * sizeof(float));

  hipMalloc((void **)&b_cuda, Size * sizeof(float));

  // copy memory to GPU
  hipMemcpy(m_cuda, m, Size * Size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(a_cuda, a, Size * Size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_cuda, b, Size * sizeof(float), hipMemcpyHostToDevice);

  int block_size, grid_size;

  block_size = MAXBLOCKSIZE;
  grid_size = (Size / block_size) + (!(Size % block_size) ? 0 : 1);
  // printf("1d grid size: %d\n",grid_size);

  dim3 dimBlock(block_size);
  dim3 dimGrid(grid_size);
  // dim3 dimGrid( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

  int blockSize2d, gridSize2d;
  blockSize2d = BLOCK_SIZE_XY;
  gridSize2d = (Size / blockSize2d) + (!(Size % blockSize2d ? 0 : 1));

  dim3 dimBlockXY(blockSize2d, blockSize2d);
  dim3 dimGridXY(gridSize2d, gridSize2d);

  for (t = 0; t < (Size - 1); t++) {
    hipLaunchKernelGGL(Fan1, dim3(dimGrid), dim3(dimBlock), 0, 0, m_cuda,
                       a_cuda, Size, t);
    hipLaunchKernelGGL(Fan2, dim3(dimGridXY), dim3(dimBlockXY), 0, 0, m_cuda,
                       a_cuda, b_cuda, Size, Size - t, t);
    checkCUDAError("Fan2");
  }
  // copy memory back to CPU
  hipMemcpy(m, m_cuda, Size * Size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(a, a_cuda, Size * Size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(b, b_cuda, Size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(m_cuda);
  hipFree(a_cuda);
  hipFree(b_cuda);
}

/*------------------------------------------------------
 ** BackSub() -- Backward substitution
 **------------------------------------------------------
 */

void BackSub() {
  // solve "bottom up"
  int i, j;
  for (i = 0; i < Size; i++) {
    finalVec[Size - i - 1] = b[Size - i - 1];
    for (j = 0; j < i; j++) {
      finalVec[Size - i - 1] -= *(a + Size * (Size - i - 1) + (Size - j - 1)) *
                                finalVec[Size - j - 1];
    }
    finalVec[Size - i - 1] =
        finalVec[Size - i - 1] / *(a + Size * (Size - i - 1) + (Size - i - 1));
  }
}

void InitMat(float *ary, int nrow, int ncol) {
  int i, j;

  for (i = 0; i < nrow; i++) {
    for (j = 0; j < ncol; j++) {
      fscanf(fp, "%f", ary + Size * i + j);
    }
  }
}

/*------------------------------------------------------
 ** PrintMat() -- Print the contents of the matrix
 **------------------------------------------------------
 */
void PrintMat(float *ary, int nrow, int ncol) {
  int i, j;

  for (i = 0; i < nrow; i++) {
    for (j = 0; j < ncol; j++) {
      printf("%8.2f ", *(ary + Size * i + j));
    }
    printf("\n");
  }
  printf("\n");
}

/*------------------------------------------------------
 ** InitAry() -- Initialize the array (vector) by reading
 ** data from the data file
 **------------------------------------------------------
 */
void InitAry(float *ary, int ary_size) {
  int i;

  for (i = 0; i < ary_size; i++) {
    fscanf(fp, "%f", &ary[i]);
  }
}

/*------------------------------------------------------
 ** PrintAry() -- Print the contents of the array (vector)
 **------------------------------------------------------
 */
void PrintAry(float *ary, int ary_size) {
  std::cerr << " Store results to output!!" << std::endl;
  // Store the result into a file.
  FILE *fpo = fopen("nat_result.txt", "w");

  int i;
  for (i = 0; i < ary_size; i++) {
    fprintf(fpo, "%.2f", ary[i]);
  }
  fclose(fpo);
}
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

void VerifyResult() {
  int i, j;
  float tmp_out = 0;
  for (i = 0; i < Size; i++) {
    for (j = 0, tmp_out = 0; j < Size; j++)
      tmp_out += (*(a + Size * i + j) * finalVec[j]);
    if (abs(tmp_out - b[i]) > 0.01) {
      printf("Test Failed\n");
      printf("out[%d]: %f; b[%d]:%f; diff:%f\n", i, tmp_out, i, b[i],
             b[i] - tmp_out);
      return;
    }
  }

  printf("Test Pass\n");
  return;
}
