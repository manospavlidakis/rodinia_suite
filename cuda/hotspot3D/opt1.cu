
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc,
        int nx, int ny, int nz,
        float ce, float cw,
        float cn, float cs,
        float ct, float cb,
        float cc)
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
            + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    return;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap,
        float Rx, float Ry, float Rz,
        float dt, int numiter)
{
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(float) * nx * ny * nz;
    float  *tIn_d, *tOut_d, *p_d;
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpy(tIn_d, tIn, s, hipMemcpyHostToDevice);
    hipMemcpy(p_d, p, s, hipMemcpyHostToDevice);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);
    for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
        float *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
#ifdef DEBUG
        float *in1 = 0, *in2 = 0, *out = 0;
        size_t sz = sizeof(float) * nx * ny * nz;
        in1 = (float *)malloc(sz);
        in2 = (float *)malloc(sz);
        out = (float *)malloc(sz);
        cudaMemcpy(in1, tIn_d, sz, cudaMemcpyDeviceToHost);
        cudaMemcpy(out, tOut_d, sz, cudaMemcpyDeviceToHost);
        cudaMemcpy(in2, p_d, sz, cudaMemcpyDeviceToHost);
        for (int i=0; i<nx*ny*nz; i++){
            std::cerr<<"i: "<<i<<" in2 = "<<in2[i]<<" in1 = "<<in1[i]<<", out = "<<out[i]<<std::endl;
        }
#endif

    }
#ifdef DEBUG
      cudaError_t err = cudaSuccess;
      err = cudaGetLastError();
      if (err != cudaSuccess){
          std::cerr<<"Error: "<<cudaGetErrorString(err)<<" err: "<<err<<std::endl;
          abort();
      }
#endif
    hipMemcpy(tOut, tOut_d, s, hipMemcpyDeviceToHost);
    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}

