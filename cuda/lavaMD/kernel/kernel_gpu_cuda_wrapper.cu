#include "./../main.h"
#include "./kernel_gpu_cuda.cu"
#include "./kernel_gpu_cuda_wrapper.h"

void kernel_gpu_cuda_wrapper(par_str par_cpu, dim_str dim_cpu, box_str *box_cpu,
                             FOUR_VECTOR *rv_cpu, fp *qv_cpu,
                             FOUR_VECTOR *fv_cpu) {
  box_str *d_box_gpu;
  FOUR_VECTOR *d_rv_gpu;
  fp *d_qv_gpu;
  FOUR_VECTOR *d_fv_gpu;

  dim3 threads;
  dim3 blocks;

  blocks.x = dim_cpu.number_boxes;
  blocks.y = 1;
  threads.x = NUMBER_THREADS;
  threads.y = 1;

  hipMalloc((void **)&d_box_gpu, dim_cpu.box_mem);
  hipMalloc((void **)&d_rv_gpu, dim_cpu.space_mem);
  hipMalloc((void **)&d_qv_gpu, dim_cpu.space_mem2);
  hipMalloc((void **)&d_fv_gpu, dim_cpu.space_mem);

  hipMemcpy(d_box_gpu, box_cpu, dim_cpu.box_mem, hipMemcpyHostToDevice);
  hipMemcpy(d_rv_gpu, rv_cpu, dim_cpu.space_mem, hipMemcpyHostToDevice);
  hipMemcpy(d_qv_gpu, qv_cpu, dim_cpu.space_mem2, hipMemcpyHostToDevice);
  hipMemcpy(d_fv_gpu, fv_cpu, dim_cpu.space_mem, hipMemcpyHostToDevice);

  // launch kernel - all boxes
  kernel_gpu_cuda<<<blocks, threads>>>(par_cpu, dim_cpu, d_box_gpu, d_rv_gpu,
                                       d_qv_gpu, d_fv_gpu);
#ifdef DEBUG
  checkCUDAError("Start");
#endif

  hipMemcpy(fv_cpu, d_fv_gpu, dim_cpu.space_mem, hipMemcpyDeviceToHost);

  hipFree(d_rv_gpu);
  hipFree(d_qv_gpu);
  hipFree(d_fv_gpu);
  hipFree(d_box_gpu);
}
